/**
 * Description: ��ȡCUDA����
 * Author: Bao Wenjie
 * Date: 2021/3/4
 */

#include <hip/hip_runtime.h>

#include <iostream>

using namespace ::std;


int main()
{
	hipError_t cudaStatus;
	int num = 0;
	cudaStatus = hipGetDeviceCount(&num);
	cout << "���У�" << num << "�� GPU" << endl << endl;
	hipDeviceProp_t prop;
	for (int i = 0; i < num; i++)
	{
		hipGetDeviceProperties(&prop, i);
		cout << "��" << i << "�� GPU" << endl;
		cout << "GPU����: " << prop.name << endl;
		cout << "�Դ��СΪ: " << prop.totalGlobalMem << endl;
		cout << "�汾��: " << prop.major << "." << prop.minor << endl;
		cout << "GPU�����: " << prop.multiProcessorCount << endl;
	}
	
}