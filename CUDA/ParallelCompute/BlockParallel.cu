#include "hip/hip_runtime.h"
/**
 * Description: �鲢��
 * Author: Bao Wenjie
 * Date: 2021/3/5
 * REF: https://blog.csdn.net/Augusdi/article/details/12833235?utm_medium=distribute.pc_relevant.none-task-blog-BlogCommendFromMachineLearnPai2-1.control&dist_request_id=&depth_1-utm_source=distribute.pc_relevant.none-task-blog-BlogCommendFromMachineLearnPai2-1.control
 */

#include "hip/hip_runtime.h"
#include ""
#include <iostream>

using namespace ::std;


hipError_t addWithCuda(int* c, const int* a, const int* b, size_t size);


__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = blockIdx.x;
    c[i] = a[i] + b[i];
}
int main()
{
    const int arraySize = 5;
    int a[arraySize], b[arraySize];
    for (int i = 0; i < arraySize; i++)
    {
        a[i] = rand();
        b[i] = rand();
    }
    int c[arraySize] = { 0 };
    
    // ���淵�صļ���״̬
    hipError_t cudaStatus;
    cudaStatus = addWithCuda(c, a, b, arraySize);
	
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
	
    for (int i = 0; i < arraySize; i++)
        cout << a[i] << " ";
    cout << endl;
    for (int i = 0; i < arraySize; i++)
        cout << b[i] << " ";
    cout << endl;
    for (int i = 0; i < arraySize; i++)
        cout << c[i] << " ";
    cout << endl;
	cudaStatus = hipDeviceReset();
	
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}



hipError_t addWithCuda(int* c, const int* a, const int* b, size_t size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // ѡ�����е��豸(GPU)
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    // ����GPU�豸���ڴ�
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
	
    // �������ݵ�GPU
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // ���к˺���
    addKernel << <size, 1 >> > (dev_c, dev_a, dev_b);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    return cudaStatus;
}