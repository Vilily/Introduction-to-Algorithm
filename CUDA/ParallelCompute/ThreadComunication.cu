#include "hip/hip_runtime.h"
/**
 * Description: �̲߳���
 * Author: Bao Wenjie
 * Date: 2021/3/5
 * REF: https://blog.csdn.net/Augusdi/article/details/12833235?utm_medium=distribute.pc_relevant.none-task-blog-BlogCommendFromMachineLearnPai2-1.control&dist_request_id=&depth_1-utm_source=distribute.pc_relevant.none-task-blog-BlogCommendFromMachineLearnPai2-1.control
 */

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <iostream>
using namespace ::std;

hipError_t addWithCuda(int* c, const int* a, size_t size);

__global__ void addKernel(int* c, const int* a)
{
    int i = threadIdx.x;
    extern __shared__ int smem[];
    smem[i] = a[i];
    __syncthreads();
    // 0���߳���ƽ����
    if (i == 0)
    {
        c[0] = 0;
        for (int d = 0; d < 5; d++)
        {
            c[0] += smem[d] * smem[d];
        }
    }
    //1���߳����ۼ�
    if (i == 1)
    {
        c[1] = 0;
        for (int d = 0; d < 5; d++)
        {
            c[1] += smem[d];
        }
    }
    //2���߳����۳�
    if (i == 2)
    {
        c[2] = 1;
        for (int d = 0; d < 5; d++)
        {
            c[2] *= smem[d];
        }
    }
}


int main()
{
    const int arraySize = 5;
    int a[arraySize];
    for (int i = 0; i < arraySize; i++)
        a[i] = rand();
    int c[arraySize] = { 0 };

	// ����
    hipError_t cudaStatus = addWithCuda(c, a, arraySize);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	// ��ӡ������
    for (int i = 0; i < arraySize; i++)
        cout << a[i] << " ";
    cout << endl;
    for (int i = 0; i < arraySize; i++)
        cout << c[i] << " ";
    cout << endl;
	
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}


hipError_t addWithCuda(int* c, const int* a, size_t size)
{
    int* dev_a = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

	// ѡ��GPU�豸
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	// ��ȡ�Դ�
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	// ���������ݿ������豸
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	// ����kernel����
    addKernel << <1, size, size * sizeof(int), 0 >> > (dev_c, dev_a);
	// �߳�ͬ��
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

	// �����ݴ��豸����������
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    return cudaStatus;
}