#include "hip/hip_runtime.h"
/**
 * Description: �̲߳���
 * Author: Bao Wenjie
 * Date: 2021/3/4
 * REF: https://blog.csdn.net/Augusdi/article/details/12833235?utm_medium=distribute.pc_relevant.none-task-blog-BlogCommendFromMachineLearnPai2-1.control&dist_request_id=&depth_1-utm_source=distribute.pc_relevant.none-task-blog-BlogCommendFromMachineLearnPai2-1.control
 */

#include <hip/hip_runtime.h>
#include <>
#include <iostream>

using namespace ::std;

hipError_t addWithCuda(int* c, const int* a, const int* b, size_t size);

// kernel����
__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

	// ���淵�صļ���״̬
    hipError_t cudaStatus;
    cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
	
    cudaStatus = hipDeviceReset();
	
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}


hipError_t addWithCuda(int* c, const int* a, const int* b, size_t size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
	
    hipError_t cudaStatus;

	// ѡ�����е��豸(GPU)
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
	
    // ����GPU�豸���ڴ�
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
	
    // �������ݵ�GPU
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	
    // ���к˺���
	addKernel <<< 1, size >>> (dev_c, dev_a, dev_b);
	cudaStatus = hipDeviceSynchronize();	//ͬ���߳�
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
	// ��GPU���ݿ�����CPU
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	
Error:
    hipFree(dev_c);	//�ͷ�GPU�豸���ڴ�
    hipFree(dev_a);
    hipFree(dev_b);
    return cudaStatus;
}