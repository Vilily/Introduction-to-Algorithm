#include "hip/hip_runtime.h"
/**
 * Description: ����GPUʵ�ּӷ�
 * Author: Bao Wenjie
 * Date: 2021/3/3
 * REF: https://blog.csdn.net/xierhacker/article/details/52473747?utm_medium=distribute.pc_relevant.none-task-blog-BlogCommendFromMachineLearnPai2-3.control&dist_request_id=&depth_1-utm_source=distribute.pc_relevant.none-task-blog-BlogCommendFromMachineLearnPai2-3.control
 */
#include <hip/hip_runtime.h>
#include <>
#include <cstdio>

using namespace ::std;
__global__ void add(int a, int b, int* c)
{
	*c = a + b;
}

int main()
{
	int c;
	int a = 1;
	int b = 7;
	int* dev_c;
	// ��GPU�Ϸ����ڴ�
	hipMalloc((void**)&dev_c, sizeof(int));
	// ������������dev
	add << <1, 1 >> > (a, b, dev_c);
	// ��GPU���ݸ��Ƶ�����
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	// �ͷ�GPU�ڴ�
	hipFree(dev_c);
	printf("%d + %d = %d\n", a, b, c);
	return 0;
}