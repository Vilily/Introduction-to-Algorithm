#include "hip/hip_runtime.h"
/**
 * Description: ����GPUʵ�־���ӷ�
 * Author: Bao Wenjie
 * Date: 2021/3/3
 */

#include <hip/hip_runtime.h>
#include <>
#include <iostream>

using namespace ::std;
const int max_x = 10;
const int max_y = 10;

__global__ void kernel_matrix_add(int* a, int* b, int* res, int maxn)
{
	res[maxn * blockIdx.x + blockIdx.y] = a[maxn * blockIdx.x + blockIdx.y] + b[maxn * blockIdx.x + blockIdx.y];
}

int main()
{
	int res[max_y * max_x], a[max_y * max_x], b[max_y * max_x];
	// �豸�ڴ��ַ
	int* dev_res, * dev_a, * dev_b;

	for (int i = 0; i < max_y * max_x; i++)
	{
		a[i] = b[i] = rand();
	}
	// ����GPU�ռ�
	hipMalloc((void**)&dev_res, max_x * max_y * sizeof(int));
	hipMalloc((void**)&dev_a, max_x * max_y * sizeof(int));
	hipMalloc((void**)&dev_b, max_x * max_y * sizeof(int));

	// ���������ݸ��Ƶ��豸
	hipMemcpy(dev_a, a, max_x * max_y * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, max_x * max_y * sizeof(int), hipMemcpyHostToDevice);

	// ���ú˺���
	dim3 x = dim3(max_y, max_x, 1);
	kernel_matrix_add << <x, 1 >> > (dev_a, dev_b, dev_res, max_x);

	// ���豸���ݸ��Ƶ�����
	hipMemcpy(res, dev_res, max_x * max_y * sizeof(int), hipMemcpyDeviceToHost);

	// �ͷſռ�
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_res);

	for (int i = 0; i < max_x * max_y; i++)
	{
		cout << a[i] << " ";
	}
	cout << endl;
	for (int i = 0; i < max_x * max_y; i++)
	{
		cout << b[i] << " ";
	}
	cout << endl;
	for (int i = 0; i < max_x * max_y; i++)
	{
		cout << res[i] << " ";
	}
	return 0;
}