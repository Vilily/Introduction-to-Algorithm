/**
 * Description: CUDA Hello World!
 * Author: Bao Wenjie
 * Date: 2021/3/3
 * REF: https://blog.csdn.net/xierhacker/article/details/52473747?utm_medium=distribute.pc_relevant.none-task-blog-BlogCommendFromMachineLearnPai2-3.control&dist_request_id=&depth_1-utm_source=distribute.pc_relevant.none-task-blog-BlogCommendFromMachineLearnPai2-3.control
 */

#include <hip/hip_runtime.h>

#include <cstdio>

int main()
{
	printf("Hello World\n");
}