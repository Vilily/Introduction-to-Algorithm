#include "hip/hip_runtime.h"
/**
 * Description: ����GPUʵ�������ӷ�
 * Author: Bao Wenjie
 * Date: 2021/3/3
 * REF: https://blog.csdn.net/xierhacker/article/details/52473747?utm_medium=distribute.pc_relevant.none-task-blog-BlogCommendFromMachineLearnPai2-3.control&dist_request_id=&depth_1-utm_source=distribute.pc_relevant.none-task-blog-BlogCommendFromMachineLearnPai2-3.control
 */

#include <hip/hip_runtime.h>
#include <>
#include <iostream>

using namespace ::std;
const int maxn = 10;

__global__ void kernel_add(int* a, int* b, int* res)
{
	res[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main()
{
	int res[maxn], a[maxn], b[maxn];
	// �豸�ڴ��ַ
	int* dev_res, * dev_a, * dev_b;

	for (int i = 0; i < maxn; i++)
		a[i] = b[i] = rand();
	// ����GPU�ռ�
	hipMalloc((void**)&dev_res, maxn * sizeof(int));
	hipMalloc((void**)&dev_a, maxn * sizeof(int));
	hipMalloc((void**)&dev_b, maxn * sizeof(int));

	// ���������ݸ��Ƶ��豸
	hipMemcpy(dev_a, a, maxn * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, maxn * sizeof(int), hipMemcpyHostToDevice);

	// ���ú˺���
	kernel_add << <maxn, 1 >> > (dev_a, dev_b, dev_res);

	// ���豸���ݸ��Ƶ�����
	hipMemcpy(res, dev_res, maxn * sizeof(int), hipMemcpyDeviceToHost);

	// �ͷſռ�
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_res);

	for(int i = 0; i < maxn; i++)
	{
		cout << a[i] << " ";
	}
	cout << endl;
	for (int i = 0; i < maxn; i++)
	{
		cout << b[i] << " ";
	}
	cout << endl;
	for (int i = 0; i < maxn; i++)
	{
		cout << res[i] << " ";
	}
	return 0;
}